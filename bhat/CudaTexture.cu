#include "hip/hip_runtime.h"
#include "CudaTexture.h"
//#include "bhat/Utils.h"
#include "hip/hip_runtime_api.h"

#define TEMPLATE_SIGN template<typename VoxelType>
#define CLASS_SIGN CudaTexture<VoxelType>

TEMPLATE_SIGN
CLASS_SIGN::CudaTexture(hipExtent extent)
{
    m_Extent = extent;
    AllocateTexture();
}

TEMPLATE_SIGN
CLASS_SIGN::CudaTexture(hipExtent extent, int normalizedCoords)
{
    m_Extent = extent;
    m_NormalizedCoords = normalizedCoords;
    AllocateTexture();
}

TEMPLATE_SIGN
CLASS_SIGN::CudaTexture(hipExtent extent, int normalizedCoords, hipChannelFormatDesc desc)
{
    m_Extent = extent;
    m_NormalizedCoords = normalizedCoords;
    m_ChannelDesc = desc;
    AllocateTexture();
}

TEMPLATE_SIGN
CLASS_SIGN::~CudaTexture()
{
    checkCudaErrors(hipDestroyTextureObject(m_TexObj));
    checkCudaErrors(hipFreeArray(m_cuArray));
}

TEMPLATE_SIGN
void CLASS_SIGN::AllocateTexture()
{
    // Allocate CUDA array in device memory
    checkCudaErrors(hipMalloc3DArray(&m_cuArray, &m_ChannelDesc, m_Extent));

    // Specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = m_cuArray;

    // Specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = m_NormalizedCoords;

    // Create texture object
    checkCudaErrors(hipCreateTextureObject(&m_TexObj, &resDesc, &texDesc, NULL));
}

TEMPLATE_SIGN
void CLASS_SIGN::LoadData(thrust::device_vector<VoxelType>& data)
{
    if (m_Extent.depth <= 1)
    {
        // Copy for to 1D/2D texture object
        checkCudaErrors(hipMemcpyToArray(m_cuArray, 0, 0, thrust::raw_pointer_cast(data.data()), data.size() * sizeof(VoxelType), hipMemcpyDeviceToDevice));
    }
    else
    {
        // Copy for to 3D texture object
        hipMemcpy3DParms copyParams = { 0 };
        copyParams.srcPtr = make_hipPitchedPtr(thrust::raw_pointer_cast(data.data()), m_Extent.width * sizeof(VoxelType), m_Extent.width, m_Extent.height);
        copyParams.dstArray = m_cuArray;
        copyParams.extent = m_Extent;
        copyParams.kind = hipMemcpyDeviceToDevice;
        checkCudaErrors(hipMemcpy3D(&copyParams));
    }
}

template class CudaTexture<float>;
template class CudaTexture<float4>;